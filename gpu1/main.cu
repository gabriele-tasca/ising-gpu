
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand.h>
#include "hiprand/hiprand_kernel.h"
#include <assert.h>

// L should be  (multiple of (THR_NUMBER - 2) ) + 2
#define L 83

const int AREA = L*L;
const int NTOT = (L-2)*(L-2);

// #define T 6.
// #define T 0.1
// #define T 2.26918531421
#define T_CYCLE_START 1.6
#define T_CYCLE_END 2.9
#define T_CYCLE_STEP 0.04

#define SINGLETEMP 2.4

int n_temps = ( T_CYCLE_END - T_CYCLE_START )/ (T_CYCLE_STEP);

#define J 1.
#define SEED 1000

struct measure_plan {
    int steps_repeat;
    int t_max_sim;
    int t_measure_wait;
    int t_measure_interval; } 
static PLAN = {
    .steps_repeat = 20,
    .t_max_sim = 200,
    .t_measure_wait = 50,
    .t_measure_interval = 20  };


// print history true/false
#define HISTORY 1

const int THR_NUMBER = 29;
const int BLOCK_NUMBER  = ( L-2)/( THR_NUMBER - 2 );
const dim3 BLOCKS( BLOCK_NUMBER, BLOCK_NUMBER );
const dim3 THREADS( THR_NUMBER, THR_NUMBER );

// average tracker struct 
struct avg_tr {
    float sum;
    float sum_squares;
    int n;
};
struct avg_tr new_avg_tr(int locn) {
    struct avg_tr a = { .sum = 0, .sum_squares = 0, .n = locn};
    return a;
}
void update_avg(struct avg_tr * tr_p, float newval) {
    tr_p->sum +=  newval;
    tr_p->sum_squares += (newval*newval);
}
__device__ static inline void dev_update_avg(struct avg_tr * tr_p, float newval) {
    tr_p->sum +=  newval;
    tr_p->sum_squares += (newval*newval);
}
float average( struct avg_tr tr) {
    return (tr.sum)/((float) tr.n) ;
}
float stdev( struct avg_tr tr) {
    return sqrt(  ( tr.sum_squares)/((float) tr.n)  -  pow(( (tr.sum)/((float) tr.n) ),2)  );
}
float variance( struct avg_tr tr) {
    return (  ( tr.sum_squares)/((float) tr.n)  -  pow(( (tr.sum)/((float) tr.n) ),2)  );
}

// RNG init kernel
__global__ void initRNG(hiprandState * const rngStates, const unsigned int seed) {
    // Determine thread ID
    int blockId = blockIdx.x+ blockIdx.y * gridDim.x;
    int tid = blockId * (blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x)+ threadIdx.x;
    // Initialise the RNG
    hiprand_init(seed, tid, 0, &rngStates[tid]);
}

// static inline float unitrand(){
//     return (float)rand() / (float)RAND_MAX;
// }
__device__ static inline float dev_unitrand( hiprandState * const rngStates, unsigned int tid ){
    hiprandState localState = rngStates[tid];
    float val = hiprand_uniform(&localState);
    rngStates[tid] = localState;
    return val;
}

void init_random(char grid[L*L]) {
    for(int x = 0; x<L; x++) {
        for(int y = 0; y<L; y++) {
            grid[x+y*L] = rand() & 1;
        }
    }
}
void init_t0(char grid[L*L]) {
    for(int x = 0; x<L; x++) { 
        for(int y = 0; y<L; y++) {
            grid[x+y*L] = 0;
        }
    }
}

void dump(char grid[L*L]) {
    for(int x = 0; x<L; x++) {
        for(int y = 0; y<L; y++) {
            // if(grid[x+y*L] == 0) printf("•");
            // else printf("◘");
            if(grid[x+y*L] == 0) printf(" ");
            else printf("█");
            // printf("%i", grid[x+y*L]);
        }
        printf("\n");
    }
    printf("\n");
}
__device__ void dev_dump(char grid[L*L]) {
    for(int x = 0; x<L; x++) {
        for(int y = 0; y<L; y++) {
            // if(grid[x+y*L] == 0) printf("•");
            // else printf("◘");
            if(grid[x+y*L] == 0) printf(" ");
            else printf("█");
            // printf("%i", grid[x+y*L]);
        }
        printf("\n");
    }
    printf("\n");
}

struct coords {
    int x;
    int y;
};
__device__ static inline coords dev_get_thread_coords() {
    struct coords thread_coords;
    // the 4 lines below are outdated haha
    // assign loc_x and loc_y so that only the inner square is covered
    // also remember that now threads are launched in blocks of 32x32,
    // but only the inner 30x30 are mapped to the grid...
    // thread_coords on each block's edge threads mean nothing and should not be read
 
    thread_coords.x =  blockIdx.x*( THR_NUMBER - 2 ) + ( threadIdx.x ) ;
    thread_coords.y =  blockIdx.y*( THR_NUMBER - 2 ) + ( threadIdx.y ) ;

    return thread_coords;
}

// can segfault 
__device__ static inline char dev_shared_grid_step(char shared_grid[THR_NUMBER*THR_NUMBER], int x, int y, int xstep, int ystep) {
    return shared_grid[(x+xstep) + (y+ystep)*THR_NUMBER];
}

// segfault if applied to an edge spin, call only on the inner THR_NUMBER-1 grid
__device__ void dev_update_spin_shared(char dev_shared_grid[ THR_NUMBER*THR_NUMBER ], int x, int y , hiprandState * const rngStates, unsigned int tid, double temperature ) {
    char s0 = dev_shared_grid[x+y*THR_NUMBER];
    char j1 = s0 ^ dev_shared_grid_step(dev_shared_grid, x, y, 1, 0);
    char j2 = s0 ^ dev_shared_grid_step(dev_shared_grid, x, y, -1, 0);
    char j3 = s0 ^ dev_shared_grid_step(dev_shared_grid, x, y, 0, 1);
    char j4 = s0 ^ dev_shared_grid_step(dev_shared_grid, x, y, 0, -1);
    float dh = (float) ( -((j1 + j2 + j3 + j4) *2 -4)*2*J );

    float p = exp(  -dh / temperature);
    float ur = dev_unitrand(rngStates, tid);

    if(ur < p ) {
        dev_shared_grid[x+y*THR_NUMBER] = !dev_shared_grid[x+y*THR_NUMBER];
    } 
}


__device__ void dev_update_grid_shared(char grid[L*L], hiprandState * const rngStates, double temperature ) {
    // the first argument here is the GLOBAL grid

    // thread coords relative to the GLOBAL grid
    struct coords glob_coords = dev_get_thread_coords();
    int glob_x = glob_coords.x;
    int glob_y = glob_coords.y;

    // Determine thread ID (for RNG)
    int blockId = blockIdx.x+ blockIdx.y * gridDim.x;
    int tid = blockId * (blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x)+ threadIdx.x;

    __shared__ char shared_grid[ THR_NUMBER*THR_NUMBER ];

    shared_grid[ threadIdx.x + threadIdx.y*THR_NUMBER ] = grid[(glob_x )+ (glob_y )*L ]; // check formulas
    __syncthreads();

    // thread coords relative to the shared grid
    int shared_x = threadIdx.x;
    int shared_y = threadIdx.y;

    // macro-checkboards
    // macro-white
    if( (blockIdx.x + blockIdx.y%2)%2 == 0 ) {
        /////////////
        // checkboards
        // update only in the inner 30x30 block of threads, because the edge threads aren't mapped to any grid spins
        if ( threadIdx.x != 0 && threadIdx.x != THR_NUMBER-1 && 
            threadIdx.y != 0 && threadIdx.y != THR_NUMBER-1 ) {
            // white
            if( (glob_x + glob_y%2)%2 == 0 ) {
                dev_update_spin_shared( shared_grid, shared_x, shared_y, rngStates, tid, temperature );
            }
        }
        __syncthreads();

        if ( threadIdx.x != 0 && threadIdx.x != THR_NUMBER-1 && 
            threadIdx.y != 0 && threadIdx.y != THR_NUMBER-1 ) {
            // black
            if( (glob_x + glob_y%2)%2 == 1 ) {
                dev_update_spin_shared( shared_grid, shared_x, shared_y, rngStates, tid, temperature );
            }
        }
        __syncthreads();

        grid[(glob_x )+ (glob_y )*L ]  = shared_grid[ threadIdx.x + threadIdx.y*THR_NUMBER ] ; // check formulas
        //////////
    }
    __syncthreads();

    // macro-black
    if( (blockIdx.x + blockIdx.y%2)%2 == 1 ) {
        //////////

        // checkboards
        // update only in the inner 30x30 block of threads, because the edge threads aren't mapped to any grid spins
        if ( threadIdx.x != 0 && threadIdx.x != THR_NUMBER-1 && 
                threadIdx.y != 0 && threadIdx.y != THR_NUMBER-1 ) {
            // white
            if( (glob_x + glob_y%2)%2 == 0 ) {
                dev_update_spin_shared( shared_grid, shared_x, shared_y, rngStates, tid, temperature );
            }
        }
        __syncthreads();

        if ( threadIdx.x != 0 && threadIdx.x != THR_NUMBER-1 && 
            threadIdx.y != 0 && threadIdx.y != THR_NUMBER-1 ) {
            // black
            if( (glob_x + glob_y%2)%2 == 1 ) {
                dev_update_spin_shared( shared_grid, shared_x, shared_y, rngStates, tid, temperature );
            }
        }
        __syncthreads();

        grid[(glob_x )+ (glob_y )*L ]  = shared_grid[ threadIdx.x + threadIdx.y*THR_NUMBER ] ; // check formulas

        //////////
    }

}


__device__ void dev_update_magnetization_tracker(char dev_grid[L*L], float * dev_single_run_avg, int * dev_partial_res ) {
    struct coords glob_coords = dev_get_thread_coords();
    int glob_x = glob_coords.x;
    int glob_y = glob_coords.y;

    if ( threadIdx.x != 0 && threadIdx.x != THR_NUMBER-1 && 
        threadIdx.y != 0 && threadIdx.y != THR_NUMBER-1 ) {
        int spin = (int) dev_grid[glob_x+glob_y*L]; 
        atomicAdd(dev_partial_res, spin );
    }
    __syncthreads();
    
    if ( blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
        float val = ( ((float)  (*dev_partial_res) *2 ) - NTOT    ) / (float) NTOT;
        /*AAAAA*/ *dev_single_run_avg += val;
        *dev_partial_res = 0;
    }
        
}

__global__ /*AAAAA*/ void dev_measure_cycle_kernel(struct measure_plan pl, char * dev_grid, hiprandState * const rngStates, float * dev_single_run_avg, int * dev_partial_res , double temperature ) {
    // INNER SIM LOOPS

    int ksim=0;
    for( ; ksim<pl.t_measure_wait; ksim++) {
        dev_update_grid_shared(dev_grid, rngStates, temperature);
    }
    // end thermalization

    for( ; ksim<pl.t_max_sim; ksim++) {
        dev_update_grid_shared(dev_grid, rngStates, temperature);

        ////////////measures
        if( ksim % pl.t_measure_interval == 0) {
            dev_update_magnetization_tracker(dev_grid, dev_single_run_avg, dev_partial_res );
        }

    }
    // END INNER SIM LOOPS
}

void parall_measure_cycle(char startgrid[L*L], struct measure_plan pl, char * dev_grid, hiprandState * const rngStates, FILE *resf, double temperature ) {



    //OUTER REP LOOP
    ////////////measures
    float n_measures_per_sim = (float) ((pl.t_max_sim - pl.t_measure_wait)/pl.t_measure_interval);
    
    struct avg_tr outer_avg_tr = new_avg_tr(pl.steps_repeat);
    

    // extra space needed by dev_update_magnetization_tracker
    int * dev_partial_res;
    hipMalloc(&dev_partial_res, sizeof(int));


    for( int krep=0; krep< pl.steps_repeat; krep++) {
        
        /*AAAAA*/ float single_run_avg = 0.;
        /*AAAAA*/ float * dev_single_run_avg;
        /*AAAAA*/ hipMalloc(&dev_single_run_avg, sizeof(float));
        /*AAAAA*/ hipMemcpy(dev_single_run_avg, &single_run_avg, sizeof(float), hipMemcpyHostToDevice);

        // printf("seeding with %i\n", SEED+krep);
        // initialize starting grid on the device for this sim
        hipMemcpy(dev_grid, startgrid, L*L*sizeof(char), hipMemcpyHostToDevice);
  
        /*AAAAA*/ dev_measure_cycle_kernel<<<BLOCKS, THREADS>>>(pl, dev_grid, rngStates, dev_single_run_avg, dev_partial_res, temperature );

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("ERROR: %s\n", hipGetErrorString(err));
        }

        // bring back results to CPU
        /*AAAAA*/ hipMemcpy(&single_run_avg, dev_single_run_avg, sizeof(float), hipMemcpyDeviceToHost);
        /*AAAAA*/ float lres = single_run_avg/(n_measures_per_sim);
        // /*AAAAA*/ float lstdev = stdev(single_run_avg);
        if (HISTORY) printf(" temperature: %f\n", temperature);
        if (HISTORY) printf("# average of simulation %i:\n %f\n", krep+1, lres);

        update_avg(&outer_avg_tr, lres);
        
        char endgrid[L*L];
        hipMemcpy(endgrid, dev_grid, L*L*sizeof(char), hipMemcpyDeviceToHost);
        if (HISTORY) dump(endgrid);
        
        /*AAAAA*/ hipFree(dev_single_run_avg);
    
    }

    // END OUTER REP LOOP
    
    ////////////measures
    fprintf(resf, "%f ", temperature);
    fprintf(resf, "%f ", average(outer_avg_tr));
    fprintf(resf, "%f\n", stdev(outer_avg_tr));
    
    hipFree(dev_partial_res);

}



int main() {
    // L should be  (multiple of THR_NUMBER -2) + 2
    assert( ((L-2)% (THR_NUMBER-2)  )== 0 );

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    FILE *resf = fopen("results.txt", "w");
    fprintf(resf, "# gpu1\n");
    fprintf(resf, "# parameters:\n# linear_size: %i\n", L);
    fprintf(resf, "# coupling: %f\n# repetitions: %i\n", J, PLAN.steps_repeat);
    fprintf(resf, "# simulation_t_max: %i\n# thermalization_time: %i\n# time_between_measurements: %i\n# base_random_seed: %i\n",  PLAN.t_max_sim, PLAN.t_measure_wait, PLAN.t_measure_interval, SEED);
    fprintf(resf, "# extra:\n# area: %i\n# active_spins_excluding_boundaries:%i\n", AREA, NTOT);
    fprintf(resf, "\n");
    fprintf(resf, "# columns: temperature - average magnetization - uncertainty \n");

    srand(SEED);

    // curand init
    // Allocate memory for RNG states
    hiprandState *d_rngStates = 0;

    hipMalloc((void **)&d_rngStates, THR_NUMBER*THR_NUMBER*BLOCK_NUMBER*BLOCK_NUMBER*sizeof(hiprandState));
    // Initialise RNG
    initRNG<<<BLOCKS, THREADS>>>(d_rngStates, SEED);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("ERROR: %s\n", hipGetErrorString(err));
    }

    // device grid
    char * dev_grid;
    hipMalloc(&dev_grid, L*L*sizeof(char));

    char startgrid[L*L];
    init_t0(startgrid); 
    // if (HISTORY) printf("starting grid:\n");
    // if (HISTORY) dump(startgrid);

    
    // // temp cycle:
    for( double kt=T_CYCLE_START; kt<T_CYCLE_END; kt+=T_CYCLE_STEP ) {
        parall_measure_cycle(startgrid, PLAN, dev_grid, d_rngStates, resf, kt);
    }

    // only 1:
    parall_measure_cycle(startgrid, PLAN, dev_grid, d_rngStates, resf, SINGLETEMP);
        

    hipFree(d_rngStates);
    hipFree(dev_grid);


    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float total_time = 0;
    hipEventElapsedTime(&total_time, start, stop);

    FILE *timef = fopen("time.txt", "w");
    int total_flips = n_temps * PLAN.steps_repeat * PLAN.t_max_sim * NTOT;
    fprintf(timef, "# total execution time (milliseconds):\n");
    fprintf(timef, "%f\n", total_time);
    fprintf(timef, "# total spin flips performed:\n");
    fprintf(timef, "%f\n", total_flips);
    fprintf(timef, "# average spin flips per millisecond:\n");
    fprintf(timef, "%f\n", ((float) total_flips  )/( (float) total_time ) );

    fclose(timef);

    fclose(resf);

}

