
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand.h>
#include "hiprand/hiprand_kernel.h"

#define L 20
const int AREA = L*L;
const int NTOT = (L-2)*(L-2);

// #define T 6.
// #define T 0.1
// #define T 2.26918
#define T 2.26918

#define J 1.

#define SEED 100



struct measure_plan {
    int steps_repeat;
    int t_max_sim;
    int t_measure_wait;
    int t_measure_interval; } 
static PLAN = {
    .steps_repeat = 1,
    .t_max_sim = 80,
    .t_measure_wait = 10,
    .t_measure_interval = 10  };


// average tracker struct 
struct avg_tr {
    float sum;
    float sum_squares;
    int n;
};
struct avg_tr new_avg_tr(int locn) {
    struct avg_tr a = { .sum = 0, .sum_squares = 0, .n = locn};
    return a;
}
void update_avg(struct avg_tr * tr_p, float newval) {
    tr_p->sum +=  newval;
    tr_p->sum_squares += (newval*newval);
}
__device__ static inline void dev_update_avg(struct avg_tr * tr_p, float newval) {
    tr_p->sum +=  newval;
    tr_p->sum_squares += (newval*newval);
}
float average( struct avg_tr tr) {
    return (tr.sum)/((float) tr.n) ;
}
float stdev( struct avg_tr tr) {
    return sqrt(  ( tr.sum_squares)/((float) tr.n)  -  pow(( (tr.sum)/((float) tr.n) ),2)  );
}
float variance( struct avg_tr tr) {
    return (  ( tr.sum_squares)/((float) tr.n)  -  pow(( (tr.sum)/((float) tr.n) ),2)  );
}

// RNG init kernel
__global__ void initRNG(hiprandState * const rngStates, const unsigned int seed) {
    // Determine thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialise the RNG
    hiprand_init(seed, tid, 0, &rngStates[tid]);
}

float unitrand(){
    return (float)rand() / (float)RAND_MAX;
}
__device__ float dev_unitrand( hiprandState * const rngStates, unsigned int tid ){
    hiprandState localState = rngStates[tid];
    return hiprand_uniform(&localState);
}

void init_random(char grid[L*L]) {
    for(int x = 0; x<L; x++) {
        for(int y = 0; y<L; y++) {
            grid[x+y*L] = rand() & 1;
        }
    }
}
void init_t0(char grid[L*L]) {
    for(int x = 0; x<L; x++) { 
        for(int y = 0; y<L; y++) {
            grid[x+y*L] = 0;
        }
    }
}


void dump(char grid[L*L]) {
    for(int x = 0; x<L; x++) {
        for(int y = 0; y<L; y++) {
            // if(grid[x+y*L] == 0) printf("•");
            // else printf("◘");
            if(grid[x+y*L] == 0) printf(" ");
            else printf("█");
            // printf("%i", grid[x+y*L]);
        }
        printf("\n");
    }
    printf("\n");
}
__device__ void dev_dump(char grid[L*L]) {
    for(int x = 0; x<L; x++) {
        for(int y = 0; y<L; y++) {
            // if(grid[x+y*L] == 0) printf("•");
            // else printf("◘");
            if(grid[x+y*L] == 0) printf(" ");
            else printf("█");
            // printf("%i", grid[x+y*L]);
        }
        printf("\n");
    }
    printf("\n");
}

// can segfault 
__device__ char dev_grid_step(char grid[L*L], int x, int y, int xstep, int ystep) {
    return grid[(x+xstep)  + (y+ystep)*L];
}

// segfault if applied to an edge spin, call only on the inner L-1 grid
__device__ void dev_update_spin(char dev_grid[L*L], int x, int y , hiprandState * const rngStates, unsigned int tid, float temperature ) {
    char s0 = dev_grid[x+y*L];
    char j1 = s0 ^ dev_grid_step(dev_grid, x, y, 1, 0);
    char j2 = s0 ^ dev_grid_step(dev_grid, x, y, -1, 0);
    char j3 = s0 ^ dev_grid_step(dev_grid, x, y, 0, 1);
    char j4 = s0 ^ dev_grid_step(dev_grid, x, y, 0, -1);
    float dh = (float) ( -((j1 + j2 + j3 + j4) *2 -4)*2*J );
    // printf("dh: %f \n", dh);

    float p = exp(  -dh / temperature);

    // remove
    hiprandState localState = rngStates[tid];
    float ur = hiprand_uniform(&localState);
    rngStates[tid] = localState;

    // float ur = dev_unitrand(rngStates, tid); 
    if (threadIdx.x == 0)  printf("p: %f, unitrand: %f \n", p, ur);
    if (threadIdx.x == 1)  printf("p: %f, unitrand: %f \n", p, ur);
    if (threadIdx.x == 2)  printf("p: %f, unitrand: %f \n", p, ur);
    if(ur < p ) {
        // printf("flipping at x%i y%i\n", x, y);
        dev_grid[x+y*L] = !dev_grid[x+y*L];
    } 
}

// for now with nthreads = NTOT
__device__ void dev_update_grid(char dev_grid[L*L], hiprandState * const rngStates, float temperature ) {
    // assign loc_x and loc_y so that only the inner square is covered
    int loc_y = (  threadIdx.x / (L-2) ) +1;
    int loc_x = (  threadIdx.x % (L-2) ) +1;
    
    // printf("%i ", loc_x);

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // int tid = threadIdx.x;    // change

    // white
    if( (loc_x + loc_y%2)%2 == 0 ) {
        // if (threadIdx.x == 6) printf("white");
        dev_update_spin( dev_grid, loc_x, loc_y, rngStates, tid, temperature );
    }
    __syncthreads();
    // black
    if( (loc_x + loc_y%2)%2 == 1 ) {
        // if (threadIdx.x == 6) printf("black");
        dev_update_spin( dev_grid, loc_x, loc_y, rngStates, tid, temperature );
    }
    __syncthreads();
    
}

float measure_m(char grid[L*L]) {
    int m = 0;
    for(int x = 1; x<L-1; x++) {
        for(int y = 1; y<L-1; y++) {
            m += (grid[x+y*L]*2. -1.);
            // printf("x %i m %f \n", x, grid[x+y*L] );
        }
    }
    return (((float) m ) / (float) NTOT) ;
}
__device__ void dev_update_magnetization_tracker(char dev_grid[L*L], struct avg_tr * dev_tr_p, float * dev_partial_res ) {
    int y = (  threadIdx.x / (L-2) ) +1;
    int x = (  threadIdx.x % (L-2) ) +1;
    float spin = (float) dev_grid[x+y*L]; 
    atomicAdd(dev_partial_res, (spin*2.)-1.  );
    __syncthreads();
    
    if (threadIdx.x == 0) {
        *dev_partial_res = *dev_partial_res / (float) NTOT;
        // printf("this this %f\n", *dev_partial_res);
        dev_update_avg( dev_tr_p, *dev_partial_res);
        *dev_partial_res = 0;
    }
    __syncthreads();
    
}

__global__ void dev_measure_cycle_kernel(struct measure_plan pl, char * dev_grid, hiprandState * const rngStates, avg_tr * dev_single_run_avg, float * dev_partial_res , float temperature ) {
    // INNER SIM LOOPS

    int ksim=0;
    for( ; ksim<pl.t_measure_wait; ksim++) {
        dev_update_grid(dev_grid, rngStates, temperature);
    }
    // end thermalization

    for( ; ksim<pl.t_max_sim; ksim++) {
        dev_update_grid(dev_grid, rngStates, temperature);
        
        if (threadIdx.x == 0) {
            printf(" time %i \n", ksim);
            dev_dump(dev_grid);
        }

        ////////////measures
        if( ksim % pl.t_measure_interval == 0) {
            dev_update_magnetization_tracker(dev_grid, dev_single_run_avg, dev_partial_res );
        }

    }
    // END INNER SIM LOOPS        
    ////////////measures
    // update_avg(&avg_of_all_sims_tr, average(sim_avg_tr));
}

void parall_measure_cycle(char startgrid[L*L], struct measure_plan pl, char * dev_grid, hiprandState * const rngStates, FILE *resf, float temperature ) {
    fprintf(resf, "# cpu1\n");
    fprintf(resf, "# parameters:\n# linear_size: %i\n", L);
    fprintf(resf, "# temperature: %f\n# temp_start: %f\n# coupling: %f\n# repetitions: %i\n", temperature, 0., J, pl.steps_repeat);
    fprintf(resf, "# simulation_t_max: %i\n# thermalization_time: %i\n# time_between_measurements: %i\n# base_random_seed: %i\n",  pl.t_max_sim, pl.t_measure_wait, pl.t_measure_interval, SEED);
    fprintf(resf, "# extra:\n# area: %i\n# active_spins_excluding_boundaries:%i\n", AREA, NTOT);


    //OUTER REP LOOP
    ////////////measures
    float n_measures_per_sim = (float) ((pl.t_max_sim - pl.t_measure_wait)/pl.t_measure_interval);
    
    struct avg_tr outer_avg_tr = new_avg_tr(pl.steps_repeat);
    

    // extra space needed by dev_update_magnetization_tracker
    float * dev_partial_res;
    hipMalloc(&dev_partial_res, sizeof(float));


    for( int krep=0; krep< pl.steps_repeat; krep++) {
        
        struct avg_tr single_run_avg = new_avg_tr(n_measures_per_sim);
        struct avg_tr * dev_single_run_avg;
        hipMalloc(&dev_single_run_avg, sizeof(struct avg_tr));
        hipMemcpy(dev_single_run_avg, &single_run_avg, sizeof(struct avg_tr), hipMemcpyHostToDevice);

        initRNG<<<1, NTOT>>>(rngStates, SEED+krep);

        // initialize starting grid on the device for this sim
        hipMemcpy(dev_grid, startgrid, L*L*sizeof(char), hipMemcpyHostToDevice);
  
        dev_measure_cycle_kernel<<<1, NTOT>>>(pl, dev_grid, rngStates, dev_single_run_avg, dev_partial_res, temperature );

        // bring back results to CPU
        hipMemcpy(&single_run_avg, dev_single_run_avg, sizeof(struct avg_tr), hipMemcpyDeviceToHost);
        float lres = average(single_run_avg);
        float lstdev = stdev(single_run_avg);
        fprintf(resf, "# average of simulation %i:\n %f +- %f\n", krep+1, lres, lstdev);
        update_avg(&outer_avg_tr, lres);

        char endgrid[L*L];
        hipMemcpy(endgrid, dev_grid, L*L*sizeof(char), hipMemcpyDeviceToHost);
        dump(endgrid);
    
    }

    // END OUTER REP LOOP
    
    ////////////measures
    fprintf(resf, "# average of all simulations: %f +- %f\n", average(outer_avg_tr), stdev(outer_avg_tr));
    


}



int main() {
    FILE *resf = fopen("results.txt", "w");

    srand(SEED);

    // curand init
    // Allocate memory for RNG states
    hiprandState *d_rngStates = 0;
    // cudaMalloc((void **)&d_rngStates, grid.x * block.x * sizeof(curandState));
    hipMalloc((void **)&d_rngStates, NTOT*sizeof(hiprandState));
    // Initialise RNG
    initRNG<<<1, NTOT>>>(d_rngStates, SEED);

    // device grid
    char * dev_grid;
    hipMalloc(&dev_grid, L*L*sizeof(char));


    char startgrid[L*L];
    init_t0(startgrid);

    dump(startgrid);



    parall_measure_cycle(startgrid, PLAN, dev_grid, d_rngStates, resf, T);

    hipFree(&d_rngStates);
    hipFree(dev_grid);

    fclose(resf);

}

