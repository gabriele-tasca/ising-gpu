
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand.h>
#include "hiprand/hiprand_kernel.h"

# define SEED 1000
# define NTOT 10

// RNG init kernel
__global__ void initRNG(hiprandState * const rngStates, const unsigned int seed) {
    // Determine thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialise the RNG
    hiprand_init(seed, tid, 0, &rngStates[tid]);
    if (threadIdx.x == 0) for(int i=0; i<1000; i++)      printf("from init rng\n");

}

__device__ static inline float dev_unitrand( hiprandState * const rngStates, unsigned int tid ){
    hiprandState localState = rngStates[tid];
    return hiprand_uniform(&localState);
}

__global__ void spam_rands(hiprandState * const rngStates, unsigned int tid ) {
    printf("le aoeufbakhfb\n");
    for(int i=0; i<10; i++) {


        float ur = dev_unitrand(rngStates, tid); 
        if (threadIdx.x == 0)  printf("unitrand: %f \n", ur);
        if (threadIdx.x == 1)  printf("unitrand: %f \n", ur);
        if (threadIdx.x == 2)  printf("unitrand: %f \n", ur);
        
    } 
}


int main() {
    // curand init
    // Allocate memory for RNG states
    hiprandState *d_rngStates = 0;
    // cudaMalloc((void **)&d_rngStates, grid.x * block.x * sizeof(curandState));
    hipMalloc((void **)&d_rngStates, NTOT*sizeof(hiprandState));
    // Initialise RNG
    initRNG<<<1, NTOT>>>(d_rngStates, SEED);
    printf("le edge\n");
    spam_rands<<<1, NTOT>>>(d_rngStates, SEED);



   
    hipFree(&d_rngStates);
}

