
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand.h>
#include "hiprand/hiprand_kernel.h"
#include <assert.h>

// L should be  (multiple of (THR_NUMBER - 2) ) + 2
const int THR_NUMBER = 30;

#define SETBLOCKNUM 7
const int L = (THR_NUMBER -2)* SETBLOCKNUM +2;

// #define MULTISPIN unsigned char
#define MULTISPIN unsigned int

const int MULTISIZE = sizeof(MULTISPIN) *8;


#define T_CYCLE_START 1.4
#define T_CYCLE_END 3.1
#define T_CYCLE_STEP 0.02

#define SINGLETEMP 3.0

int n_temps = ( T_CYCLE_END - T_CYCLE_START )/ (T_CYCLE_STEP);

#define J 1.

#define SEED 1000

const int AREA = L*L;
const int NTOT = (L-2)*(L-2);
// static const float EXP4_TRESHOLD = exp( -(4.*J) / T);
// static const float EXP8_TRESHOLD = exp( -(8.*J) / T);

#define STEPS_REPEAT 3
#define T_MAX_SIM 600
#define T_MEASURE_WAIT 50
#define T_MEASURE_INTERVAL 10

// print history true/false
#define HISTORY 0

const int BLOCK_NUMBER  = ( L-2)/( THR_NUMBER - 2 );
const dim3 BLOCKS( BLOCK_NUMBER, BLOCK_NUMBER );
const dim3 THREADS( THR_NUMBER, THR_NUMBER );


// average tracker struct
struct avg_tr {
    float sum;
    float sum_squares;
    int n;
};
struct avg_tr new_avg_tr(int locn) {
    struct avg_tr a = { .sum = 0, .sum_squares = 0, .n = locn};
    return a;
}

void update_avg(struct avg_tr * tr_p, float newval) {
    tr_p->sum +=  newval;
    tr_p->sum_squares += (newval*newval);
}
float average( struct avg_tr tr) {
    return (tr.sum)/((float) tr.n) ;
}
float stdev( struct avg_tr tr) {
    return sqrt(  ( tr.sum_squares)/((float) tr.n)  -  pow(( (tr.sum)/((float) tr.n) ),2)  );
}
// float variance( struct avg_tr tr) {
//     return (  ( tr.sum_squares)/((float) tr.n)  -  pow(( (tr.sum)/((float) tr.n) ),2)  );
// }

//multispin averages, hard-coded to track a number MULTISPIN * STEPS_REPEAT of values
struct multiavg_tr {
    float sum[MULTISIZE * STEPS_REPEAT];
    float sum_squares[MULTISIZE * STEPS_REPEAT];
    int n; // number of terms in the avg
};
// localn is not multisize*steps_repeat, it's the number of terms that will contribute to each avg ...
struct multiavg_tr new_multiavg_tr(int localn) {
    struct multiavg_tr a;
    for(int k=0; k<MULTISIZE * STEPS_REPEAT; k++ ) {
        a.sum[k] = 0.;
        a.sum_squares[k] = 0.;
    }
    a.n = localn;
    return a;
}
// must be 0 =< k <MULTISIZE * STEPS_REPEAT
// void update_multiavg(struct multiavg_tr * tr_p, float newval, int k) {
//     tr_p->sum[k] +=  newval;
//     tr_p->sum_squares[k] += (newval*newval);
// }
// __device__ void dev_update_multiavg(struct multiavg_tr * tr_p, float newval, int k) {
//     tr_p->sum[k] +=  newval;
//     tr_p->sum_squares[k] += (newval*newval);
// }
float multiaverage( struct multiavg_tr tr, int k) {
    return (tr.sum[k])/((float) tr.n) ;
}
float multistdev( struct multiavg_tr tr, int k) {
    return sqrt(  ( tr.sum_squares[k])/((float) tr.n)  -  pow(( (tr.sum[k])/((float) tr.n) ),2)  );
}
// float multivariance( struct multiavg_tr tr, int k) {
//     return (  ( tr.sum_squares[k])/((float) tr.n)  -  pow(( (tr.sum[k])/((float) tr.n) ),2)  );
// }

// RNG init kernel
__global__ void initRNG(hiprandState * const rngStates, const int seed) {
    // Determine thread ID
    int blockId = blockIdx.x+ blockIdx.y * gridDim.x;
    int tid = blockId * (blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x)+ threadIdx.x;
    // Initialise the RNG
    hiprand_init(seed, tid, 0, &rngStates[tid]);
}


struct coords {
    int x;
    int y;
};
__device__ coords dev_get_thread_coords() {
    struct coords thread_coords;
 
    thread_coords.x =  blockIdx.x*( THR_NUMBER - 2 ) + ( threadIdx.x ) ;
    thread_coords.y =  blockIdx.y*( THR_NUMBER - 2 ) + ( threadIdx.y ) ;

    return thread_coords;
}

// float unitrand(){
//     return (float)rand() / (float)RAND_MAX;
// }
__device__ float dev_unitrand( hiprandState * const rngStates, unsigned int tid ){
    hiprandState localState = rngStates[tid];
    float val = hiprand_uniform(&localState);
    rngStates[tid] = localState;
    return val;
}

// index has to be less that MULTISIZE
__device__ void dev_set_spin_1 (MULTISPIN * multi, int index) {
    *multi |= 1 << index;
}
__device__ void dev_set_spin_0 (MULTISPIN * multi, int index) {
    *multi &= ~(1 << index);
}
__device__ MULTISPIN dev_read_spin(MULTISPIN multi, int index) {
    // return (( multi >> ((MULTISPIN) index ) ) & ((MULTISPIN) 1));
    // if (multi & (1 << index) == 0) {
    //     return 0;
    // } else {
    //     return 1;
    // }
    return ( (multi >> index) & 1 );
}
// each bit exp8 and exp8 describes the Metropolis RNG result for that bit,
// specifying if the random r is bigger or smaller than the relevant values e^(4J/kT) and e^(8J/kT) (passed from outside)
__device__ MULTISPIN generate_exp4_mask(float exp4, float exp8, hiprandState * const rngStates, int tid ) {
    MULTISPIN res;
    for(int k=0; k<MULTISIZE; k++) {
        float random_number = dev_unitrand(rngStates, tid); 
        if( exp4 > random_number && random_number > exp8) { // this is taken from the article and works. the version below might not but slightly simplifies some things
        // if( exp4 > random_number) {
            dev_set_spin_1(&res, k);
        } else {
            dev_set_spin_0(&res, k);
        }
    }
    return res;
}
__device__ MULTISPIN generate_exp8_mask(float exp8, hiprandState * const rngStates, int tid ) {
    MULTISPIN res;
    for(int k=0; k<MULTISIZE; k++) {
        float random_number = dev_unitrand(rngStates, tid); 
        if( random_number < exp8 ) {
            dev_set_spin_1(&res, k);
        } else {
            dev_set_spin_0(&res, k);
        }
    }
    return res;
}

MULTISPIN init_random_multispin() {
    return (MULTISPIN) rand(); // just spam random bits
}
void init_random_grid(MULTISPIN grid[L*L]) {
    for(int x = 0; x<L; x++) {
        for(int y = 0; y<L; y++) {
            grid[x+y*L] = init_random_multispin();
        }
    }
}


MULTISPIN init_t0_multispin() {
    return (MULTISPIN) 0; // should be all zeros for all sensible multispin types
}
void init_t0_grid(MULTISPIN grid[L*L]) {
    for(int x = 0; x<L; x++) {
        for(int y = 0; y<L; y++) {
            grid[x+y*L] = init_t0_multispin();
        }
    }
}

// void flip(MULTISPIN grid[L*L], int x, int y) {
//     grid[x+y*L] = ~grid[x+y*L];
// }

// can segfault 
__device__ static inline MULTISPIN dev_shared_grid_step(MULTISPIN shared_grid[THR_NUMBER*THR_NUMBER], int x, int y, int xstep, int ystep) {
    return shared_grid[(x+xstep) + (y+ystep)*THR_NUMBER];
}


// segfault if applied to an edge spin, must be called only on the inner L-1 grid
__device__ void dev_update_multispin_shared(MULTISPIN grid[THR_NUMBER*THR_NUMBER], int x, int y, float exp4, float exp8, hiprandState * const rngStates, int tid ) {

    MULTISPIN s0 = grid[x+y*THR_NUMBER];

    MULTISPIN exp4_mask = generate_exp4_mask(exp4, exp8, rngStates, tid ); // here
    MULTISPIN exp8_mask = generate_exp8_mask(exp8, rngStates, tid );

    // "energy variables" indicating whether s0 is equal or opposite to each of its 4 neighbours 
    MULTISPIN i1 = s0 ^ dev_shared_grid_step(grid, x, y, 1, 0);
    MULTISPIN i2 = s0 ^ dev_shared_grid_step(grid, x, y, -1, 0);
    MULTISPIN i3 = s0 ^ dev_shared_grid_step(grid, x, y, 0, 1);
    MULTISPIN i4 = s0 ^ dev_shared_grid_step(grid, x, y, 0, -1);
    
    // bit sums with carry over between the i variables
    MULTISPIN j1 = i1 & i2;
    MULTISPIN j2 = i1 ^ i2;
    MULTISPIN j3 = i3 & i4;
    MULTISPIN j4 = i3 ^ i4;

    // logic for deciding whether to flip s0 or not
    MULTISPIN flip_mask = ( ((j1 | j3) | (~(j1^j3) & (j2&j4))  )  |   ((j2 | j4) & exp4_mask  )   |   exp8_mask );

    grid[x+y*THR_NUMBER] = grid[x+y*THR_NUMBER] ^ flip_mask;


    // explanation:
    // spins | i1234 | deltaE | j1 j2  j3 j4 |
    //   1   |   1   |        |              |    
    //  101  |  1 1  |   -8   |  1 0    1 0  |                   
    //   1   |   1   |        |              |
                                                            
    //   0   |   0   |        |              |    
    //  101  |  1 1  |   -4   |  0 1    1 0  |         (j1 | j3)          
    //   1   |   1   |        |              |
                                                            
    //   0   |   0   |        |  0 0    1 0  |    
    //  001  |  0 1  |    0   |      or      |-------------------------                  
    //   1   |   1   |        |  0 1    0 1  |      ~(j1^j3) & (j2&j4))
    // ------------------------------------------------------------------
                                                           
    //   0   |   0   |        |              |    
    //  000  |  0 0  |    +4  |              |       (j2 | j4) & exp4      
    //   1   |   1   |        |              |
    // ------------------------------------------------------------------ 
                                                           
    //   0   |   0   |        |              |    
    //  000  |  0 0  |    +8  |  0 0    0 0  |           exp8       
    //   0   |   0   |        |              |

    // the first 2 cases are detected by (j1 | j3) and lead to the spin flip regardless of the RNG roll.
    // the deltaH = 0 case can result in two different forms for the j's depending on ho the spins are paired. 
    //   the first of these is correctly picked up by (j1 | j3), while the second needs its own expression ~(j1^j3) & (j2&j4))
    // in the 4th case, detected by (j2 | j4), the spin is flipped only if the RNG roll is lucky enough (exp4 = 1)
    // if we still haven't flipped, we get to the last case. here the spin is flipped only if the RNG roll gives the luckiest result (exp8 = 1).
    
}



// non GPU function
void multidump_first(MULTISPIN grid[L*L]) {
    // printf("first bit grid (out of %i):\n", MULTISIZE);
    for(int x = 0; x<L; x++) {
        for(int y = 0; y<L; y++) {
            
            if(( grid[x+y*L] & 1 ) == 0) printf(" ");
            else printf("█");

        }
        printf("\n");
    }
    printf("\n");
}

// non GPU function
void multidump_a_few(MULTISPIN grid[L*L]) {
    for(int k=0; k<5; k++) {
        printf("grid on bit %i (out of %i):\n", k, MULTISIZE);
        for(int x = 0; x<L; x++) {
            for(int y = 0; y<L; y++) {
                
                if(( grid[x+y*L] & (1 << k) ) == 0) printf(" ");
                else printf("█");

            }
            printf("\n");
        }
        printf("\n");
    }
}


__global__ void dev_measure_cycle_kernel(MULTISPIN * dev_grid, hiprandState * const rngStates, float * dev_single_run_avgs, int * dev_partial_res, float exp4, float exp8, int ksim ) {


        // setup

        struct coords glob_coords = dev_get_thread_coords();
        int glob_x = glob_coords.x;
        int glob_y = glob_coords.y;
    
        // Determine thread ID (for RNG)
        int blockId = blockIdx.x+ blockIdx.y * gridDim.x;
        int tid = blockId * (blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x)+ threadIdx.x;
    
        __shared__ MULTISPIN shared_grid[ THR_NUMBER*THR_NUMBER ];
        shared_grid[ threadIdx.x + threadIdx.y*THR_NUMBER ] = dev_grid[(glob_x )+ (glob_y )*L ];
        __syncthreads();
        
        __shared__ int blocksum[ MULTISIZE ];
        
        if ( threadIdx.x == 0 && threadIdx.y == 0 ) {
            for (int multik=0; multik<MULTISIZE; multik++) {
                blocksum[ multik ] = 0;
            }
        }

        __syncthreads();

        ////////////////////////////////////////////
        ////// measure magnetization
        ////////////////////////////////////////////
        if(ksim > T_MEASURE_WAIT && ksim % T_MEASURE_INTERVAL == 0) {
        // this condition does not depend on the thread id in any way
            for (int multik=0; multik<MULTISIZE; multik++) {
                
                if ( threadIdx.x != 0 && threadIdx.x != THR_NUMBER-1 
                    && threadIdx.y != 0 && threadIdx.y != THR_NUMBER-1 ) {
                    int lspin = (int) dev_read_spin(shared_grid[threadIdx.x + threadIdx.y*THR_NUMBER], multik );
                    atomicAdd(  &(blocksum[ multik ]), lspin  ); // change with pointer arithm?
                }
                __syncthreads();
                if ( threadIdx.x == 0 && threadIdx.y == 0 ) {
                    int blockntot = (THR_NUMBER-2)*(THR_NUMBER-2);
                    float nval = ((float) ( blocksum[ multik] *2 - blockntot ))/ ( (float) blockntot );
                    atomicAdd(&(dev_single_run_avgs[multik]), nval);
                    blocksum[ multik  ] = 0;
                }



            }
            // if ( threadIdx.y == 1 && threadIdx.x == 1 ) {
            //     for(int multik=0; multik <MULTISIZE; multik++) {
            //         dev_partial_res[multik] = 0;
            //     }
            //     printf(" devpart before %i\n", dev_partial_res[0]);
                
            // }
            // __syncthreads();

            




        }

        __syncthreads();
    
        ////////////////////////////////////////////
        ////// update
        ////////////////////////////////////////////
        // macro-checkboards
        // macro-white
        if( (blockIdx.x + blockIdx.y%2)%2 == 0 ) {
            /////////////
            // checkboards
            // update only in the inner 30x30 block of threads, because the edge threads aren't mapped to any grid spins
            if ( threadIdx.x != 0 && threadIdx.x != THR_NUMBER-1 && 
                threadIdx.y != 0 && threadIdx.y != THR_NUMBER-1 ) {
                // white
                if( (glob_x + glob_y%2)%2 == 0 ) {
                    dev_update_multispin_shared( shared_grid, threadIdx.x, threadIdx.y, exp4, exp8, rngStates, tid);
                }
            }
            __syncthreads();
    
            if ( threadIdx.x != 0 && threadIdx.x != THR_NUMBER-1 && 
                threadIdx.y != 0 && threadIdx.y != THR_NUMBER-1 ) {
                // black
                if( (glob_x + glob_y%2)%2 == 1 ) {
                    dev_update_multispin_shared( shared_grid, threadIdx.x, threadIdx.y, exp4, exp8, rngStates, tid);
                }
            }
            __syncthreads();
    
            // if ( threadIdx.x > 0 && threadIdx.x != THR_NUMBER-1 && 
                // threadIdx.y > 0 && threadIdx.y != THR_NUMBER-1 ) {
                // dev_grid[(glob_x )+ (glob_y )*L ]  = shared_grid[ threadIdx.x + threadIdx.y*THR_NUMBER ] ; 
            // }
            //////////
        }
        __syncthreads();
    
        // macro-black
        if( (blockIdx.x + blockIdx.y%2)%2 == 1 ) {
            //////////
    
            // checkboards
            // update only in the inner 30x30 block of threads, because the edge threads aren't mapped to any grid spins
            if ( threadIdx.x != 0 && threadIdx.x != THR_NUMBER-1 && 
                    threadIdx.y != 0 && threadIdx.y != THR_NUMBER-1 ) {
                // white
                if( (glob_x + glob_y%2)%2 == 0 ) {
                    dev_update_multispin_shared( shared_grid, threadIdx.x, threadIdx.y, exp4, exp8, rngStates, tid);
                }
            }
            __syncthreads();
    
            if ( threadIdx.x != 0 && threadIdx.x != THR_NUMBER-1 && 
                threadIdx.y != 0 && threadIdx.y != THR_NUMBER-1 ) {
                // black
                if( (glob_x + glob_y%2)%2 == 1 ) {
                    dev_update_multispin_shared( shared_grid, threadIdx.x, threadIdx.y, exp4, exp8, rngStates, tid);
                }
            }
            __syncthreads();
    
        }
        
        if ( threadIdx.x > 0 && threadIdx.x != THR_NUMBER-1 && 
            threadIdx.y > 0 && threadIdx.y != THR_NUMBER-1 ) {
            dev_grid[(glob_x )+ (glob_y )*L ]  = shared_grid[ threadIdx.x + threadIdx.y*THR_NUMBER ] ; 
        }
        //////////




    

        // __syncthreads();
    

}

void parall_measure_cycle(MULTISPIN startgrid[L*L], MULTISPIN * dev_grid, float exp4, float exp8, hiprandState * const rngStates, FILE *resf) {


    float n_measures_per_sim = (float) ((T_MAX_SIM - T_MEASURE_WAIT)/T_MEASURE_INTERVAL);

    //OUTER REP LOOP  
    // struct multiavg_tr single_run_avgs = new_multiavg_tr(n_measures_per_sim);
    // struct multiavg_tr * dev_single_run_avgs;
    // cudaMalloc(&dev_single_run_avgs, sizeof(struct multiavg_tr));
    // cudaMemcpy(dev_single_run_avgs, &single_run_avgs, sizeof(struct multiavg_tr), cudaMemcpyHostToDevice);

    // extra space needed by update_magnetization
    // int * dev_partial_res;
    // cudaMalloc(&dev_partial_res, sizeof(int));

    float single_run_avgs[MULTISIZE];
    for (int k=0; k<MULTISIZE; k++) {single_run_avgs[k] = 0.;}
    float * dev_single_run_avgs;
    hipMalloc(&dev_single_run_avgs, MULTISIZE*sizeof(float));
    hipMemcpy(dev_single_run_avgs, &single_run_avgs, MULTISIZE*sizeof(float), hipMemcpyHostToDevice);

    // extra space needed by update_magnetization
    int partial_res[MULTISIZE];
    for (int k=0; k<MULTISIZE; k++) {partial_res[k] = 0;}
    int * dev_partial_res;
    hipMalloc(&dev_partial_res, MULTISIZE*sizeof(int));
    hipMemcpy(dev_partial_res, &partial_res, MULTISIZE*sizeof(int), hipMemcpyHostToDevice);


    // outer average
    struct avg_tr avg_of_runs = new_avg_tr( MULTISIZE * STEPS_REPEAT );

    for( int krep=0; krep< STEPS_REPEAT; krep++) {
        if (HISTORY) printf("# simulation %i\n", krep+1);
        if (HISTORY) printf("#    waiting thermalization for the first %i sim steps.\n", T_MEASURE_WAIT);

        hipMemcpy(dev_grid, startgrid, L*L*sizeof(MULTISPIN), hipMemcpyHostToDevice);
        
        // kernel
        for (int ksim=0; ksim<T_MAX_SIM; ksim++) {

            dev_measure_cycle_kernel<<<BLOCKS, THREADS>>>(dev_grid, rngStates, dev_single_run_avgs, dev_partial_res, exp4, exp8, ksim );
        }
            hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("kernel: ERROR: %s\n", hipGetErrorString(err));
        } else printf("kernel: no ERROR: %s\n", hipGetErrorString(err));

        // results
        hipMemcpy(&single_run_avgs, dev_single_run_avgs, MULTISIZE*sizeof(float), hipMemcpyDeviceToHost);
        for(int multik=0; multik <MULTISIZE; multik++) {
            float lres = single_run_avgs[multik] / (n_measures_per_sim * BLOCK_NUMBER*BLOCK_NUMBER); // change
            if (HISTORY) printf("# average on bit %i\n: %f\n", multik+1, lres);
            update_avg(&avg_of_runs, lres);
            // reset averages
            single_run_avgs[multik] = 0.;
            partial_res[multik] = 0;
        }
        hipMemcpy(dev_single_run_avgs, &single_run_avgs, MULTISIZE*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_partial_res, &   partial_res, MULTISIZE*sizeof(int), hipMemcpyHostToDevice);


        if (HISTORY) printf("# end simulation %i\n", krep+1);
    }
    // END OUTER REPETITION LOOP

    float l2av =  average(avg_of_runs);
    float l2stdev =  stdev(avg_of_runs);
    if (HISTORY) printf("# overall average \n: %f +- %f\n", l2av, l2stdev);
    fprintf(resf, "%f ", l2av);
    fprintf(resf, "%f\n", l2stdev);

    // cudaMemcpy(&single_run_avgs, dev_single_run_avgs, sizeof(struct multiavg_tr), cudaMemcpyDeviceToHost);
    
    // ///////////////
    // struct avg_tr avg_of_runs = new_avg_tr( MULTISIZE * STEPS_REPEAT );
    // for(int k=0; k <MULTISIZE * STEPS_REPEAT; k++) {
    //     float lres = multiaverage(single_run_avgs, k);
    //     float lstdev = multistdev(single_run_avgs, k);

    //     if (HISTORY) printf("# average of simulation %i\n: %f +- %f\n", k+1, lres, lstdev);
    //     update_avg(&avg_of_runs, lres);
    // }
    // float l2av =  average(avg_of_runs);
    // float l2stdev =  stdev(avg_of_runs);
    // if (HISTORY) printf("# overall average \n: %f +- %f\n", l2av, l2stdev);
    // fprintf(resf, "%f ", l2av);
    // fprintf(resf, "%f\n", l2stdev);
    // ////////////////

    // grid for displaying end-state (of last rep only)
    MULTISPIN endgrid[L*L];
    hipMemcpy(endgrid, dev_grid, L*L*sizeof(MULTISPIN), hipMemcpyDeviceToHost);

    if (HISTORY) multidump_first(endgrid);

    hipFree(dev_partial_res);
    hipFree(dev_single_run_avgs);

}



int main() {
    // L should be  (multiple of THR_NUMBER -2) + 2
    assert( ((L-2)% (THR_NUMBER-2)  )== 0 );

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    FILE *resf = fopen("results.txt", "w");
    fprintf(resf, "# gpu1\n");
    fprintf(resf, "# parameters:\n# linear_size: %i\n", L);
    fprintf(resf, "# coupling: %f\n# repetitions: %i\n", J, STEPS_REPEAT);
    fprintf(resf, "# simulation_t_max: %i\n# thermalization_time: %i\n# time_between_measurements: %i\n# base_random_seed: %i\n",  T_MAX_SIM,T_MEASURE_WAIT, T_MEASURE_INTERVAL, SEED);
    fprintf(resf, "# extra:\n# area: %i\n# active_spins_excluding_boundaries:%i\n", AREA, NTOT);
    fprintf(resf, "\n");
    fprintf(resf, "# columns: temperature - average magnetization - uncertainty \n");
    
    // still used for init_random_grid
    srand(SEED);

    // curand init
    // Allocate memory for RNG states
    hiprandState *d_rngStates = 0;

    hipMalloc((void **)&d_rngStates, THR_NUMBER*THR_NUMBER*BLOCK_NUMBER*BLOCK_NUMBER*sizeof(hiprandState));
    // Initialise RNG
    initRNG<<<BLOCKS, THREADS>>>(d_rngStates, SEED);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("initRNG: ERROR: %s\n", hipGetErrorString(err));
    } else printf("initRNG: no ERROR: %s\n", hipGetErrorString(err));
     

    // device grid
    MULTISPIN * dev_grid;
    hipMalloc(&dev_grid, L*L*sizeof(MULTISPIN));

    // original grid on the cpu
    MULTISPIN startgrid[L*L];
    init_t0_grid(startgrid);
    // multidump_a_few(startgrid);

    // // temp cycle:
    for( float kt=T_CYCLE_START; kt<T_CYCLE_END; kt+=T_CYCLE_STEP ) {
        const float EXP4 = exp( -(4.*J) / kt);
        const float EXP8 = exp( -(8.*J) / kt);
        fprintf(resf, "%f ", kt);
        if (HISTORY) printf("temperature: %f\n", kt);
        parall_measure_cycle(startgrid, dev_grid, EXP4, EXP8, d_rngStates, resf);
    }

    // // // // only 1:
    // // // // just one:
    // const float EXP4 = exp( -(4.*J) / SINGLETEMP);
    // const float EXP8 = exp( -(8.*J) / SINGLETEMP);
    // fprintf(resf, "%f ", SINGLETEMP);
    // if (HISTORY) printf("temperature: %f\n", SINGLETEMP);
    // parall_measure_cycle(startgrid, dev_grid, EXP4, EXP8, d_rngStates, resf);
    
    // printf(" ERROR? rng malloc size: %i\n", THR_NUMBER*THR_NUMBER*BLOCK_NUMBER*BLOCK_NUMBER*sizeof(curandState));
    // printf(" ERROR? shared memory used: %i\n", THR_NUMBER*THR_NUMBER*sizeof(MULTISPIN) + BLOCK_NUMBER*BLOCK_NUMBER*MULTISIZE*sizeof(int));

    hipFree(d_rngStates);
    hipFree(dev_grid);


    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float total_time = 0;
    hipEventElapsedTime(&total_time, start, stop);

    FILE *timef = fopen("time.txt", "w");
    long int total_flips = ((long int)(n_temps))* ((long int)((STEPS_REPEAT))) * ((long int)(T_MAX_SIM)) * ((long int)(MULTISIZE)) * ((long int)(NTOT));
    
    fprintf(timef, "# gpu1\n");
    fprintf(timef, "# total execution time (milliseconds):\n");
    fprintf(timef, "%f\n", total_time);
    fprintf(timef, "# total spin flips performed:\n");
    fprintf(timef, "%li\n", total_flips);
    fprintf(timef, "# average spin flips per millisecond:\n");
    fprintf(timef, "%Lf\n", ((long double) total_flips  )/( (long double) total_time ) );

    fclose(timef);

    fclose(resf);

    
    return 0;
}

